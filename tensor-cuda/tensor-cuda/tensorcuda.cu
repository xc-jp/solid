#include <hip/hip_runtime.h>
#include <cstdio>
#include "tensorcuda.cuh"

#define checkCudaErr(...)        \
  {                              \
    int err = (int) __VA_ARGS__; \
    if (err)                     \
      return err;                \
  }

int tensor_cuda::devMalloc(size_t bytes, void **p) {
  checkCudaErr(hipMalloc(p, bytes));
  return 0;
}

int tensor_cuda::devFree(void *devp) {
  checkCudaErr(hipFree(devp));
  return 0;
}

int tensor_cuda::memcpyToDev(size_t bytes, void *devDst, void *hostSrc) {
  checkCudaErr(hipMemcpy(devDst, hostSrc, bytes, hipMemcpyHostToDevice));
  return 0;
}

int tensor_cuda::memcpyFromDev(size_t bytes, void *hostDst, void *devSrc) {
  checkCudaErr(hipMemcpy(hostDst, devSrc, bytes, hipMemcpyDeviceToHost));
  return 0;
}
