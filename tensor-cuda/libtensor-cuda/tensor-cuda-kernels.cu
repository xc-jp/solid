#include <hip/hip_runtime.h>
#include "tensor-cuda.h"

#define checkCudaErr(...)       \
  {                             \
    int err = (int)__VA_ARGS__; \
    if (err)                    \
      return err;               \
  }

int deviceSynchronize() {
  return hipDeviceSynchronize();
}

int devMalloc(const size_t bytes, void** p) {
  checkCudaErr(hipMalloc(p, bytes));
  return 0;
}

int devFree(void* devp) {
  checkCudaErr(hipFree(devp));
  return 0;
}

int memcpyToDev(const size_t bytes, void* devDst, void* hostSrc) {
  checkCudaErr(hipMemcpy(devDst, hostSrc, bytes, hipMemcpyHostToDevice));
  return 0;
}

int memcpyFromDev(const size_t bytes,
                               void* hostDst,
                               void* devSrc) {
  checkCudaErr(hipMemcpy(hostDst, devSrc, bytes, hipMemcpyDeviceToHost));
  return 0;
}

int getDevice(int* device) {
  checkCudaErr(hipGetDevice(device));
  return 0;
}
