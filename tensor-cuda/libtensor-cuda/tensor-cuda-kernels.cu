#include <hip/hip_runtime.h>
#include <cstdio>
#include "tensor-cuda-kernels.cuh"

#define checkCudaErr(...)       \
  {                             \
    int err = (int)__VA_ARGS__; \
    if (err)                    \
      return err;               \
  }

int tensor_cuda::devMalloc(const size_t bytes, void** p) {
  checkCudaErr(hipMalloc(p, bytes));
  return 0;
}

int tensor_cuda::devFree(void* devp) {
  checkCudaErr(hipFree(devp));
  return 0;
}

int tensor_cuda::memcpyToDev(const size_t bytes, void* devDst, void* hostSrc) {
  checkCudaErr(hipMemcpy(devDst, hostSrc, bytes, hipMemcpyHostToDevice));
  return 0;
}

int tensor_cuda::memcpyFromDev(const size_t bytes,
                               void* hostDst,
                               void* devSrc) {
  checkCudaErr(hipMemcpy(hostDst, devSrc, bytes, hipMemcpyDeviceToHost));
  return 0;
}
